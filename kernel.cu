#include "hip/hip_runtime.h"
﻿
#pragma OPENCL EXTENSION cl_amd_printf : enable

typedef struct _CAMERA
{
	float3 position;
	float3 direction;
	float3 right;
	float3 top;
	float focalLength;
	float aspect;
} CAMERA;

typedef struct _RAY
{
	float3 position;
	float3 direction;
} RAY;

typedef struct _BOX
{
	float3 minimum;
	float3 maximum;
} BOX;

RAY getRay(global CAMERA *camera, float x, float y)
{
	RAY ray;
	ray.position = camera->position.xyz;
	ray.direction = 
		camera->direction * camera->focalLength + 
		camera->right * 400 * x * camera->aspect + 
		camera->top * 400 * y;
	ray.direction = normalize(ray.direction);
	return ray;
}

bool intersect_ray_box(RAY *ray, BOX *box)
{
	float dist = 0.0f;
	float maxValue = FLT_MAX;
	if (fabs(ray->direction.x) < 1E-06f)
	{
		if ((ray->position.x < box->minimum.x) || (ray->position.x > box->maximum.x))
		{
			return false;
		}
	}
	else
	{
		float num11 = 1.0f / ray->direction.x;
		float num8 = (box->minimum.x - ray->position.x) * num11;
		float num7 = (box->maximum.x - ray->position.x) * num11;
		if (num8 > num7)
		{
			float temp = num8;
			num8 = num7;
			num7 = temp;
		}
		dist = max(num8, dist);
		maxValue = min(num7, maxValue);
		if (dist > maxValue)
		{
			return false;
		}
	}
	if (fabs(ray->direction.y) < 1E-06f)
	{
		if ((ray->position.y < box->minimum.y) || (ray->position.y > box->maximum.y))
		{
			return false;
		}
	}
	else
	{
		float num10 = 1.0f / ray->direction.y;
		float num6 = (box->minimum.y - ray->position.y) * num10;
		float num5 = (box->maximum.y - ray->position.y) * num10;
		if (num6 > num5)
		{
			float temp = num6;
			num6 = num5;
			num5 = temp;
		}
		dist = max(num6, dist);
		maxValue = min(num5, maxValue);
		if (dist > maxValue)
		{
			return false;
		}
	}
	if (fabs(ray->direction.z) < 1E-06f)
	{
		if ((ray->position.z < box->minimum.z) || (ray->position.z > box->maximum.z))
		{
			return false;
		}
	}
	else
	{
		float num9 = 1.0f / ray->direction.z;
		float num4 = (box->minimum.z - ray->position.z) * num9;
		float num3 = (box->maximum.z - ray->position.z) * num9;
		if (num4 > num3)
		{
			float temp = num4;
			num4 = num3;
			num3 = temp;
		}
		dist = max(num4, dist);
		maxValue = min(num3, maxValue);
		if (dist > maxValue)
		{
			return false;
		}
	}
	return true;
}

__kernel void render(	global uchar4 *renderTarget, 
						int width, 
						int height, 
						global CAMERA *camera,
						global uchar *world,
						int sizeX,
						int sizeY,
						int sizeZ,
						uchar threshold)
{
	const int x = get_global_id(0);
	const int y = get_global_id(1);
	if(x >= width || y >= height)
		return;
	int id = x + width * y;
	
	const float pixx = 2.0f * ((float)x / width) - 1.0f;
	const float pixy = 2.0f * ((float)y / height) - 1.0f;
	
	BOX volume;
	RAY ray = getRay(camera, pixx, pixy);

	volume.minimum = (float3)(0);
	volume.maximum = (float3)(sizeX, sizeY, sizeZ);
	
	renderTarget[id] = (uchar4)(255, 128, 64, 255);
	if(!intersect_ray_box(&ray, &volume))
	{
		return;
	}
	
	float clip_far = 1600;
	float stepWidth = clip_far / 16000.0f;

	float3 start = ray.position;
	float3 step = sign(ray.direction);
	float3 boundary = start;
	boundary.x += step.x > 0;
	boundary.y += step.y > 0;
	boundary.z += step.z > 0;

	float3 tmax = (boundary - ray.position) / ray.direction;
	if(isnan(tmax.x)) tmax.x = FLT_MAX;
	if(isnan(tmax.y)) tmax.y = FLT_MAX;
	if(isnan(tmax.z)) tmax.z = FLT_MAX;

	float3 tdelta = step / ray.direction;
	if(isnan(tdelta.x)) tdelta.x = FLT_MAX;
	if(isnan(tdelta.y)) tdelta.y = FLT_MAX;
	if(isnan(tdelta.z)) tdelta.z = FLT_MAX;

	int px = (int)(start.x + 0.5f);
	int py = (int)(start.y + 0.5f);
	int pz = (int)(start.z + 0.5f);

	for(int i = 0; i < 750; i++)
	{
		if(px >= 0 && py >= 0 && pz >= 0 && px < sizeX && py < sizeY && pz < sizeZ)
		{
			int offset = sizeX * sizeY * pz + sizeX * py + px;
			if(world[offset] >= threshold)
			{
				renderTarget[id].x = world[offset];
				renderTarget[id].y =  world[offset];
				renderTarget[id].z =  world[offset];
				renderTarget[id].w = 255;
				return;
			}
		}
		if(tmax.x < tmax.y && tmax.x < tmax.z)
		{
			px += step.x;
			tmax.x += tdelta.x;
		}
		else if(tmax.y < tmax.z)
		{
			py += step.y;
			tmax.y += tdelta.y;
		}
		else
		{
			pz += step.z;
			tmax.z += tdelta.z;
		}
	}
}